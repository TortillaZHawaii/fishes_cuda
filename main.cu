#include "hip/hip_runtime.h"
/* Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>

#include "boid.cuh"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

#define BOID_COUNT 16384
#define BOID_POS_SIZE (BOID_COUNT * 2)

#define THREADS_PER_BLOCK 1024

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

// boids
BoidSoA d_boids;

bool isActive = true;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 1.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";

float randFloatInRange(float min, float max)
{
    float random = ((float)rand()) / (float)RAND_MAX;
    float diff = max - min;
    float r = random * diff;
    return min + r;
}

void randomizeBoids()
{
    BoidSoA h_boids;

    h_boids.positionsX = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.positionsY = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.positionsZ = (float*)malloc(sizeof(float) * BOID_COUNT);

    h_boids.velocitiesX = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.velocitiesY = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.velocitiesZ = (float*)malloc(sizeof(float) * BOID_COUNT);

    h_boids.headingsX = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.headingsY = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.headingsZ = (float*)malloc(sizeof(float) * BOID_COUNT);

    if(h_boids.positionsX == NULL || h_boids.positionsY == NULL || h_boids.positionsZ == NULL ||
       h_boids.velocitiesX == NULL || h_boids.velocitiesY == NULL || h_boids.velocitiesZ == NULL ||
       h_boids.headingsX == NULL || h_boids.headingsY == NULL || h_boids.headingsZ == NULL)
    {
        printf("Error allocating memory for boids\n");
        exit(EXIT_FAILURE);
    }

    const float max_velocity = 0.2f;

    for(int i = 0; i < BOID_COUNT; i++)
    {
        h_boids.positionsX[i] = randFloatInRange(-1.0f, 1.0f);
        h_boids.positionsY[i] = randFloatInRange(-1.0f, 1.0f);
        h_boids.positionsZ[i] = randFloatInRange(-1.0f, 1.0f);

        h_boids.velocitiesX[i] = randFloatInRange(-max_velocity, max_velocity);
        h_boids.velocitiesY[i] = randFloatInRange(-max_velocity, max_velocity);
        h_boids.velocitiesZ[i] = randFloatInRange(-max_velocity, max_velocity);

        float3 heading = make_float3(h_boids.velocitiesX[i], h_boids.velocitiesY[i], h_boids.velocitiesZ[i]);
        heading = normalize(heading);

        h_boids.headingsX[i] = heading.x;
        h_boids.headingsY[i] = heading.y;
        h_boids.headingsZ[i] = heading.z;
    }

    checkCudaErrors(hipMemcpy(d_boids.positionsX, h_boids.positionsX, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_boids.positionsY, h_boids.positionsY, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_boids.positionsZ, h_boids.positionsZ, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_boids.velocitiesX, h_boids.velocitiesX, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_boids.velocitiesY, h_boids.velocitiesY, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_boids.velocitiesZ, h_boids.velocitiesZ, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_boids.headingsX, h_boids.headingsX, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_boids.headingsY, h_boids.headingsY, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_boids.headingsZ, h_boids.headingsZ, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));

    free(h_boids.positionsX);
    free(h_boids.positionsY);
    free(h_boids.positionsZ);

    free(h_boids.velocitiesX);
    free(h_boids.velocitiesY);
    free(h_boids.velocitiesZ);

    free(h_boids.headingsX);
    free(h_boids.headingsY);
    free(h_boids.headingsZ);
}

void createBoids()
{
    checkCudaErrors(hipMalloc(&d_boids.headingsX, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&d_boids.headingsY, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&d_boids.headingsZ, sizeof(float) * BOID_COUNT));
    
    checkCudaErrors(hipMalloc(&d_boids.positionsX, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&d_boids.positionsY, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&d_boids.positionsZ, sizeof(float) * BOID_COUNT));

    hipMemset(d_boids.headingsX, 0, sizeof(float) * BOID_COUNT);
    hipMemset(d_boids.headingsY, 0, sizeof(float) * BOID_COUNT);
    hipMemset(d_boids.headingsZ, 0, sizeof(float) * BOID_COUNT);

    hipMemset(d_boids.positionsX, 0, sizeof(float) * BOID_COUNT);
    hipMemset(d_boids.positionsY, 0, sizeof(float) * BOID_COUNT);
    hipMemset(d_boids.positionsZ, 0, sizeof(float) * BOID_COUNT);

    checkCudaErrors(hipMalloc(&d_boids.velocitiesX, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&d_boids.velocitiesY, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&d_boids.velocitiesZ, sizeof(float) * BOID_COUNT));

    randomizeBoids();
}



void freeBoids()
{
    hipFree(d_boids.headingsX);
    hipFree(d_boids.headingsY);
    hipFree(d_boids.headingsZ);

    hipFree(d_boids.positionsX);
    hipFree(d_boids.positionsY);
    hipFree(d_boids.positionsZ);

    hipFree(d_boids.velocitiesX);
    hipFree(d_boids.velocitiesY);
    hipFree(d_boids.velocitiesZ);
}

void launch_kernel(BoidSoA boidsoa, float4 *pos, float time)
{
    // execute the kernel
    int boidCount = BOID_COUNT;
    int blocksCount = boidCount / THREADS_PER_BLOCK;

    steerBoid<<<blocksCount, THREADS_PER_BLOCK>>>(boidsoa, pos, time, boidCount);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    printf("\n");

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    // background color (navy blue)
    glClearColor(0.0, 0.0, 0.1, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);
    createBoids();

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    // First initialize OpenGL context, so we can properly set the GL for CUDA.
    // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    if (false == initGL(&argc, argv))
    {
        return false;
    }

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutCloseFunc(cleanup);

    // create VBO
    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    // run the cuda part
    runCuda(&cuda_vbo_resource);

    // start rendering mainloop
    glutMainLoop();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    if(isActive)
    {
        float dt = 0.015f;
        launch_kernel(d_boids, dptr, dt);
    }

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = BOID_POS_SIZE * sizeof(float4);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    // color fish
    glColor3f(1.0, 1.0, 1.0);
    // draw fish (head & tail) as line
    glDrawArrays(GL_LINES, 0,  BOID_POS_SIZE);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }

    freeBoids();
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            glutDestroyWindow(glutGetWindow());
            return;
        case ' ':
            isActive = !isActive;
            return;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        float *data = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}
