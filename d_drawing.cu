#include "hip/hip_runtime.h"
#include "d_drawing.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>
#include "defines.cuh"
#include "floatmath.cuh"

void randomizeBoids(BoidSoA* boids);

// allocates memory for boids on GPU and initializes them with random values
void createBoids(BoidSoA *boids)
{
    checkCudaErrors(hipMalloc(&boids->headingsX, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&boids->headingsY, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&boids->headingsZ, sizeof(float) * BOID_COUNT));
    
    checkCudaErrors(hipMalloc(&boids->positionsX, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&boids->positionsY, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&boids->positionsZ, sizeof(float) * BOID_COUNT));

    checkCudaErrors(hipMemset(boids->headingsX, 0, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMemset(boids->headingsY, 0, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMemset(boids->headingsZ, 0, sizeof(float) * BOID_COUNT));

    checkCudaErrors(hipMemset(boids->positionsX, 0, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMemset(boids->positionsY, 0, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMemset(boids->positionsZ, 0, sizeof(float) * BOID_COUNT));

    checkCudaErrors(hipMalloc(&boids->velocitiesX, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&boids->velocitiesY, sizeof(float) * BOID_COUNT));
    checkCudaErrors(hipMalloc(&boids->velocitiesZ, sizeof(float) * BOID_COUNT));

    randomizeBoids(boids);
}

// generates random values for boids on CPU and copies them to GPU
void randomizeBoids(BoidSoA* boids)
{
    BoidSoA h_boids;

    h_boids.positionsX = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.positionsY = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.positionsZ = (float*)malloc(sizeof(float) * BOID_COUNT);

    h_boids.velocitiesX = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.velocitiesY = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.velocitiesZ = (float*)malloc(sizeof(float) * BOID_COUNT);

    h_boids.headingsX = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.headingsY = (float*)malloc(sizeof(float) * BOID_COUNT);
    h_boids.headingsZ = (float*)malloc(sizeof(float) * BOID_COUNT);

    if(h_boids.positionsX == NULL || h_boids.positionsY == NULL || h_boids.positionsZ == NULL ||
       h_boids.velocitiesX == NULL || h_boids.velocitiesY == NULL || h_boids.velocitiesZ == NULL ||
       h_boids.headingsX == NULL || h_boids.headingsY == NULL || h_boids.headingsZ == NULL)
    {
        printf("Error allocating memory for boids\n");
        exit(EXIT_FAILURE);
    }

    const float max_velocity = 0.2f;

    for(int i = 0; i < BOID_COUNT; i++)
    {
        h_boids.positionsX[i] = randFloatInRange(-1.0f, 1.0f);
        h_boids.positionsY[i] = randFloatInRange(-1.0f, 1.0f);
        h_boids.positionsZ[i] = randFloatInRange(-1.0f, 1.0f);

        h_boids.velocitiesX[i] = randFloatInRange(-max_velocity, max_velocity);
        h_boids.velocitiesY[i] = randFloatInRange(-max_velocity, max_velocity);
        h_boids.velocitiesZ[i] = randFloatInRange(-max_velocity, max_velocity);

        float3 heading = make_float3(h_boids.velocitiesX[i], h_boids.velocitiesY[i], h_boids.velocitiesZ[i]);
        heading = normalize(heading);

        h_boids.headingsX[i] = heading.x;
        h_boids.headingsY[i] = heading.y;
        h_boids.headingsZ[i] = heading.z;
    }

    checkCudaErrors(hipMemcpy(boids->positionsX, h_boids.positionsX, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(boids->positionsY, h_boids.positionsY, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(boids->positionsZ, h_boids.positionsZ, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(boids->velocitiesX, h_boids.velocitiesX, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(boids->velocitiesY, h_boids.velocitiesY, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(boids->velocitiesZ, h_boids.velocitiesZ, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(boids->headingsX, h_boids.headingsX, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(boids->headingsY, h_boids.headingsY, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(boids->headingsZ, h_boids.headingsZ, sizeof(float) * BOID_COUNT, hipMemcpyHostToDevice));

    free(h_boids.positionsX);
    free(h_boids.positionsY);
    free(h_boids.positionsZ);

    free(h_boids.velocitiesX);
    free(h_boids.velocitiesY);
    free(h_boids.velocitiesZ);

    free(h_boids.headingsX);
    free(h_boids.headingsY);
    free(h_boids.headingsZ);
}

// frees GPU memory
void freeBoids(BoidSoA* boids)
{
    checkCudaErrors(hipFree(boids->headingsX));
    checkCudaErrors(hipFree(boids->headingsY));
    checkCudaErrors(hipFree(boids->headingsZ));

    checkCudaErrors(hipFree(boids->positionsX));
    checkCudaErrors(hipFree(boids->positionsY));
    checkCudaErrors(hipFree(boids->positionsZ));

    checkCudaErrors(hipFree(boids->velocitiesX));
    checkCudaErrors(hipFree(boids->velocitiesY));
    checkCudaErrors(hipFree(boids->velocitiesZ));
}